#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <TNtuple.h>
#include <TH1D.h>
#include <TH2F.h>
#include <TH1.h>
#include "signals.h"
#include "nll.h"

// macro to print error and abort on cuda errors, c/o stan
#define CUDA_CHECK_ERROR(call) do { \
  hipError_t err = call; \
  if (hipSuccess != err) { \
    fprintf(stderr, "Cuda error in file '%s' in line %i: %s.\n", \
            __FILE__, __LINE__, hipGetErrorString(err)); \
    exit(EXIT_FAILURE); \
  } \
} while (0)


__global__ void ll(const float* lut, const float* pars, const size_t ne,
                   const size_t ns, double* sums) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  sums[idx] = 0;
  for (int i=idx; i<(int)ne; i+=gridDim.x*blockDim.x) {
    double s = 0;
    for (size_t j=0; j<ns; j++) {
      s += pars[j] * lut[i*ns+j];
    }
    sums[idx] += log(s);
  }
}


NLL::NLL(const std::vector<Signal>& signals, TNtuple* data) {
  this->nsignals = signals.size();
  this->nevents = data->GetEntries();

  this->expectations = new double[nsignals];
  this->constraints = new double[nsignals];

  for (size_t i=0; i<nsignals; i++) {
    this->expectations[i] = signals[i].nexpected;
    this->constraints[i] = signals[i].constraint;
  }

  this->lut = NLL::build_lut(signals, data);

  CUDA_CHECK_ERROR(hipMalloc(&this->lut_device,
                              this->nevents * this->nsignals * sizeof(float)));

  CUDA_CHECK_ERROR(hipMemcpy(this->lut_device, lut,
                              this->nevents * this->nsignals * sizeof(float),
                              hipMemcpyHostToDevice));

  // pre-allocate buffers for the normalizations and output sums,
  // which change on every call
  this->normalizations = new float[nsignals];
  CUDA_CHECK_ERROR(hipMalloc(&this->normalizations_device,
                              nsignals * sizeof(float)));


  this->blocksize = 256;
  this->nblocks = 16;
  this->nthreads = this->nblocks * this->blocksize;
  this->sums = new double[this->nthreads];
  CUDA_CHECK_ERROR(hipMalloc(&this->sums_device, nthreads * sizeof(double)));
}


NLL::~NLL() {
  delete this->lut;
  delete[] this->expectations;
  delete[] this->constraints;
  delete[] this->normalizations;
  delete[] this->sums;

  CUDA_CHECK_ERROR(hipFree(this->lut_device));
  CUDA_CHECK_ERROR(hipFree(this->normalizations_device));
  CUDA_CHECK_ERROR(hipFree(this->sums_device));
}


float* NLL::build_lut(const std::vector<Signal>& signals, TNtuple* data) {
  std::cout << "NLL::build_lut: Building P(x) lookup table" << std::endl;
  int nevents = data->GetEntries();
  float* lut = new float[signals.size() * nevents];

  std::vector<float> minima;
  for (size_t i=0; i<signals.size(); i++) {
    minima.push_back(signals[i].histogram->GetMinimum(0) * 0.0001);
  }

  float e;
  float r;
  data->SetBranchAddress("e", &e);
  data->SetBranchAddress("r", &r);

  for (int i=0; i<nevents; i++) {
    data->GetEntry(i);
    for (size_t j=0; j<signals.size(); j++) {
      double v = 0;
      if (signals[j].histogram->IsA() == TH2F::Class()) {
        v = dynamic_cast<TH2F*>(signals[j].histogram)->Interpolate(r, e);
      }
      else if (signals[j].histogram->IsA() == TH1D::Class()) {
        v = dynamic_cast<TH1D*>(signals[j].histogram)->Interpolate(e);
      }
      else {
        std::cerr << "build_lut: Unknown histogram class "
                  << signals[j].histogram->ClassName() << std::endl;
        assert(false);
      }

      if (v <= 0) {
        v = minima[j];
      }
      lut[i * signals.size() + j] = v;
    }
  }

  return lut;
}


double NLL::operator()(float* norms) {
  double result = 0;

  // N + fractional constraints
  for (size_t i=0; i<this->nsignals; i++) {
    if (norms[i] < 0) {
      return 1e10;
    }
    result += norms[i];
    if (this->constraints[i] > 0) {
      result += 0.5 * pow((norms[i]/this->expectations[i] - 1) /
                          this->constraints[i], 2);
    }
  }

  CUDA_CHECK_ERROR(hipMemcpy(this->normalizations_device, norms,
                              this->nsignals * sizeof(float),
                              hipMemcpyHostToDevice));

  ll<<<this->nblocks, this->blocksize>>>(this->lut_device,
                                         this->normalizations_device,
                                         this->nevents, this->nsignals,
                                         sums_device);

  CUDA_CHECK_ERROR(hipDeviceSynchronize());

  CUDA_CHECK_ERROR(hipMemcpy(this->sums, this->sums_device,
                              this->nthreads * sizeof(double),
                              hipMemcpyDeviceToHost));

  double sum = 0;
  for (size_t i=0; i<this->nthreads; i++) {
    sum += sums[i];
  }

  result -= sum;

  return result;
}

